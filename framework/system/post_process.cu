#include "hip/hip_runtime.h"
#include "post_process.cuh"
#include <>

// namespace Pupil {
// namespace cuda {
// CUDA_INLINE CUDA_DEVICE float3 ACESToneMapping(float3 color, float adapted_lum) noexcept {
//     const float A = 2.51f;
//     const float B = 0.03f;
//     const float C = 2.43f;
//     const float D = 0.59f;
//     const float E = 0.14f;

//     color *= adapted_lum;
//     return (color * (A * color + B)) / (color * (C * color + D) + E);
// }
// CUDA_INLINE CUDA_DEVICE float3 GammaCorrection(float3 color, float gamma) {
//     return make_float3(powf(color.x, 1.f / gamma), powf(color.y, 1.f / gamma), powf(color.z, 1.f / gamma));
// }

// CUDA_GLOBAL void ACESToneMapWithGammaCorrection(float4 *output_image, const float4 *input_image, uint2 size, float gamma) {
//     int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
//     int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
//     if (pixel_x >= size.x) return;
//     if (pixel_y >= size.y) return;
//     int pixel = pixel_x + size.x * pixel_y;
//     float3 color = make_float3(input_image[pixel]);
//     color = ACESToneMapping(color, 1.f);
//     color = GammaCorrection(color, gamma);
//     output_image[pixel] = make_float4(color, input_image[pixel].w);
// }

// CUDA_GLOBAL void ACESToneMapWithoutGammaCorrection(float4 *output_image, const float4 *input_image, uint2 size) {
//     int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
//     int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
//     if (pixel_x >= size.x) return;
//     if (pixel_y >= size.y) return;
//     int pixel = pixel_x + size.x * pixel_y;
//     float3 color = make_float3(input_image[pixel]);
//     color = ACESToneMapping(color, 1.f);
//     output_image[pixel] = make_float4(color, input_image[pixel].w);
// }

// CUDA_GLOBAL void OnlyGammaCorrection(float4 *output_image, const float4 *input_image, uint2 size, float gamma) {
//     int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
//     int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
//     if (pixel_x >= size.x) return;
//     if (pixel_y >= size.y) return;
//     int pixel = pixel_x + size.x * pixel_y;
//     float3 color = make_float3(input_image[pixel]);
//     color = GammaCorrection(color, gamma);
//     output_image[pixel] = make_float4(color, input_image[pixel].w);
// }

// void PostProcess(hipStream_t stream, hipEvent_t finished_event,
//                  float4 *output_image, const float4 *input_image,
//                  uint2 size, float gamma, unsigned int post_process_type) {
//     constexpr int block_size_x = 32;
//     constexpr int block_size_y = 32;
//     int grid_size_x = (size.x + block_size_x - 1) / block_size_x;
//     int grid_size_y = (size.y + block_size_y - 1) / block_size_y;
//     dim3 grid(grid_size_x, grid_size_y);
//     dim3 block(block_size_x, block_size_y);
//     if (post_process_type == 1) {
//         ACESToneMapWithGammaCorrection<<<grid, block, 0, stream>>>(
//             output_image, input_image, size, gamma);
//     } else if (post_process_type == 2) {
//         ACESToneMapWithoutGammaCorrection<<<dim3(grid_size_x, grid_size_y),
//                                             dim3(block_size_x, block_size_y),
//                                             0, stream>>>(
//             output_image, input_image, size);
//     }
//     /*switch (post_process_type) {
//         case 0:
//             break;
//         case 1: {
//             ACESToneMapWithGammaCorrection<<<grid, block, 0, stream>>>(
//                 output_image, input_image, size, gamma);

//         } break;
//         case 2: {
//             ACESToneMapWithoutGammaCorrection<<<dim3(grid_size_x, grid_size_y),
//                                                 dim3(block_size_x, block_size_y),
//                                                 0, stream>>>(
//                 output_image, input_image, size);
//         } break;
//         case 3: {
//             OnlyGammaCorrection<<<dim3(grid_size_x, grid_size_y),
//                                   dim3(block_size_x, block_size_y),
//                                   0, stream>>>(
//                 output_image, input_image, size, gamma);
//         } break;
//     }*/
//     hipEventRecord(finished_event, stream);
// }
// }
// }// namespace Pupil::cuda

#include <iostream>
__global__ void my_kernel() { /* ... */
}

void PostProcessXX() {
    my_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "Kernel launched!" << std::endl;
}
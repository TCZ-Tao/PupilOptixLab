#include "hip/hip_runtime.h"
#include <optix.h>
#include "type.h"

#include "optix_util/util.h"
#include "optix_util/geometry.h"

#include "cuda_util/random.h"

extern "C" {
__constant__ OptixLaunchParams optix_launch_params;
}

struct PathPayloadRecord {
    float3 radiance;
    cuda::Random random;

    float3 throughput;

    float3 hit_p;
    float3 wi;
    float bsdf_sampled_pdf;

    unsigned int depth;
    bool done;
};

extern "C" __global__ void __raygen__main() {
    // const RayGenData *sbt_data = (RayGenData *)optixGetSbtDataPointer();
    const uint3 index = optixGetLaunchIndex();
    const unsigned int w = optix_launch_params.config.frame.width;
    const unsigned int h = optix_launch_params.config.frame.height;
    const unsigned int pixel_index = index.y * w + index.x;
    auto &camera = *optix_launch_params.camera.operator->();

    // optix_launch_params.frame_buffer[pixel_index] =
    //     make_float4(
    //         (float)index.x / w,
    //         (float)index.y / h, 0.f, 1.f);

    PathPayloadRecord record{};
    uint32_t u0, u1;
    optix_util::PackPointer(&record, u0, u1);

    record.done = false;
    record.depth = 0u;
    record.throughput = make_float3(1.f);
    record.radiance = make_float3(0.f);
    record.bsdf_sampled_pdf = 0.f;
    record.random.Init(4, pixel_index, optix_launch_params.frame_cnt);

    const float2 subpixel_jitter = make_float2(record.random.Next(), record.random.Next());

    const float2 subpixel =
        make_float2(
            (static_cast<float>(index.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(index.y) + subpixel_jitter.y) / static_cast<float>(h));
    // const float2 subpixel = make_float2((static_cast<float>(index.x)) / w, (static_cast<float>(index.y)) / h);
    const float4 point_on_film = make_float4(subpixel, 0.f, 1.f);

    float4 d = make_float4(
        dot(camera.sample_to_camera.r0, point_on_film),
        dot(camera.sample_to_camera.r1, point_on_film),
        dot(camera.sample_to_camera.r2, point_on_film),
        dot(camera.sample_to_camera.r3, point_on_film));

    d /= d.w;
    d.w = 0.f;
    d = normalize(d);

    float3 ray_direction = (make_float3(
        dot(camera.camera_to_world.r0, d),
        dot(camera.camera_to_world.r1, d),
        dot(camera.camera_to_world.r2, d)));

    float3 ray_origin = make_float3(
        camera.camera_to_world.r0.w,
        camera.camera_to_world.r1.w,
        camera.camera_to_world.r2.w);

    optixTrace(optix_launch_params.handle,
               ray_origin, ray_direction,
               0.001f, 1e16f, 0.f,
               255, OPTIX_RAY_FLAG_NONE,
               0, 2, 0,
               u0, u1);

    while (record.depth < optix_launch_params.config.max_depth - 1) {
        // while (record.depth < 0) {
        if (record.done)
            break;

        ray_origin = record.hit_p;
        ray_direction = record.wi;
        ++record.depth;

        optixTrace(optix_launch_params.handle,
                   ray_origin, ray_direction,
                   0.001f, 1e16f, 0.f,
                   255, OPTIX_RAY_FLAG_NONE,
                   0, 2, 0,
                   u0, u1);

        double rr = record.depth > 2 ? 0.95 : 1.0;
        if (record.random.Next() > rr)
            break;
        record.throughput /= rr;
    }

    if (optix_launch_params.frame_cnt > 0) {
        const float t = 1.f / (optix_launch_params.frame_cnt + 1.f);
        const float3 pre = make_float3(optix_launch_params.accum_buffer[pixel_index]);
        record.radiance = lerp(pre, record.radiance, t);
    }
    optix_launch_params.accum_buffer[pixel_index] = make_float4(record.radiance, 1.f);

    float3 color = optix_util::ACESToneMapping(record.radiance, 1.f);
    // color = optix_util::GammaCorrection(color, 2.2f);
    optix_launch_params.frame_buffer[pixel_index] = make_float4(color, 1.f);
}

extern "C" __global__ void __miss__default() {
    auto record = optix_util::GetPRD<PathPayloadRecord>();
    // if (optix_launch_params.env) {
    //     // TODO: environment texture
    //     float2 tex = make_float2(0.f, 0.f);
    //     record->radiance += record->throughput * optix_launch_params.env->Sample(tex);
    // }
    record->done = true;
}
extern "C" __global__ void __miss__shadow() {
    // optixSetPayload_0(0u);
}
extern "C" __global__ void __closesthit__default() {
    const HitGroupData *sbt_data = (HitGroupData *)optixGetSbtDataPointer();
    auto record = optix_util::GetPRD<PathPayloadRecord>();

    const auto ray_dir = optixGetWorldRayDirection();
    const auto ray_o = optixGetWorldRayOrigin();

    auto hit_geo = sbt_data->geo.GetHitLocalGeometry(ray_dir, sbt_data->mat.twosided);
    record->hit_p = hit_geo.position;

    if (sbt_data->emitter_index_offset >= 0) {
        unsigned int emitter_index = sbt_data->emitter_index_offset + optixGetPrimitiveIndex();
        auto &emitter = optix_launch_params.emitters[emitter_index];
        auto emission = emitter.radiance.Sample(hit_geo.texcoord);
        if (record->depth == 0) {
            record->radiance += emission;
        } else {
            float distance = length(ray_o - hit_geo.position);
            optix_util::Emitter::LocalRecord emitter_local = emitter.GetLocalInfo(hit_geo.position);
            float LNoL = dot(emitter_local.normal, -ray_dir);
            if (LNoL > 0.f) {
                float light_pdf = distance * distance / (LNoL * emitter.area) * emitter.select_probability;
                float mis = optix_util::MISWeight(record->bsdf_sampled_pdf, light_pdf);
                record->radiance += record->throughput * emission * mis;
            }
        }
    }

    auto &emitter = optix_util::Emitter::SelectOneEmiiter(record->random.Next(), optix_launch_params.emitters);
    auto emitter_local = emitter.SampleDirect(record->random.Next(), record->random.Next());
    float distance = length(emitter_local.position - hit_geo.position);
    float3 L = normalize(emitter_local.position - hit_geo.position);
    float NoL = dot(hit_geo.normal, L);
    float LNoL = dot(emitter_local.normal, -L);

    if (NoL > 0.f && LNoL > 0.f) {
        unsigned int occluded = 0u;
        optixTrace(optix_launch_params.handle,
                   hit_geo.position, L,
                   0.001f, distance - 0.001f, 0.f,
                   255, OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                   1, 2, 1, occluded);
        if (occluded == 0u) {
            float light_pdf = distance * distance / (LNoL * emitter.area) * emitter.select_probability;
            float3 wi = optix_util::ToLocal(L, hit_geo.normal);
            float3 wo = optix_util::ToLocal(-ray_dir, hit_geo.normal);
            auto [f, pdf] = sbt_data->mat.Eval(wi, wo, hit_geo.texcoord);
            if (!optix_util::IsZero(f) && pdf > 0.f) {
                float mis = optix_util::MISWeight(light_pdf, pdf);
                record->radiance += record->throughput * emitter_local.radiance * f * NoL * mis / light_pdf;
            }
        }
    }

    float2 xi = make_float2(record->random.Next(), record->random.Next());
    float3 wo = optix_util::ToLocal(-ray_dir, hit_geo.normal);
    auto [f, wi, pdf] = sbt_data->mat.Sample(xi, wo, hit_geo.texcoord);
    if (optix_util::IsZero(f * abs(wi.z)) || optix_util::IsZero(pdf))
        record->done = true;
    else {
        record->throughput *= f * abs(wi.z) / pdf;
        record->bsdf_sampled_pdf = pdf;
        record->wi = optix_util::ToWorld(wi, hit_geo.normal);
    }
}
extern "C" __global__ void __closesthit__shadow() {
    optixSetPayload_0(1u);
}
#include "hip/hip_runtime.h"
#include <optix.h>
#include "type.h"

#include "optix_util/util.h"
#include "optix_util/geometry.h"

#include "cuda_util/random.h"

extern "C" {
__constant__ OptixLaunchParams optix_launch_params;
}

struct HitInfo {
    optix_util::LocalGeometry geo;
    optix_util::material::Material mat;
    int emitter_index;
};

struct PathPayloadRecord {
    float3 radiance;
    cuda::Random random;

    float3 throughput;

    HitInfo hit;

    unsigned int depth;
    bool done;
};

extern "C" __global__ void __raygen__main() {
    // const RayGenData *sbt_data = (RayGenData *)optixGetSbtDataPointer();
    const uint3 index = optixGetLaunchIndex();
    const unsigned int w = optix_launch_params.config.frame.width;
    const unsigned int h = optix_launch_params.config.frame.height;
    const unsigned int pixel_index = index.y * w + index.x;
    auto &camera = *optix_launch_params.camera.operator->();

    // optix_launch_params.frame_buffer[pixel_index] =
    //     make_float4(
    //         (float)index.x / w,
    //         (float)index.y / h, 0.f, 1.f);

    PathPayloadRecord record{};
    uint32_t u0, u1;
    optix_util::PackPointer(&record, u0, u1);

    record.done = false;
    record.depth = 0u;
    record.throughput = make_float3(1.f);
    record.radiance = make_float3(0.f);
    record.random.Init(4, pixel_index, optix_launch_params.frame_cnt);

    const float2 subpixel_jitter = make_float2(record.random.Next(), record.random.Next());

    const float2 subpixel =
        make_float2(
            (static_cast<float>(index.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(index.y) + subpixel_jitter.y) / static_cast<float>(h));
    // const float2 subpixel = make_float2((static_cast<float>(index.x)) / w, (static_cast<float>(index.y)) / h);
    const float4 point_on_film = make_float4(subpixel, 0.f, 1.f);

    float4 d = make_float4(
        dot(camera.sample_to_camera.r0, point_on_film),
        dot(camera.sample_to_camera.r1, point_on_film),
        dot(camera.sample_to_camera.r2, point_on_film),
        dot(camera.sample_to_camera.r3, point_on_film));

    d /= d.w;
    d.w = 0.f;
    d = normalize(d);

    float3 ray_direction = normalize(make_float3(
        dot(camera.camera_to_world.r0, d),
        dot(camera.camera_to_world.r1, d),
        dot(camera.camera_to_world.r2, d)));

    float3 ray_origin = make_float3(
        camera.camera_to_world.r0.w,
        camera.camera_to_world.r1.w,
        camera.camera_to_world.r2.w);

    optixTrace(optix_launch_params.handle,
               ray_origin, ray_direction,
               0.001f, 1e16f, 0.f,
               255, OPTIX_RAY_FLAG_NONE,
               0, 2, 0,
               u0, u1);

    int depth = 0;
    auto local_hit = record.hit;

    while (true) {
        if (record.done)
            break;

        if (depth == 0) {
            if (record.hit.emitter_index >= 0) {
                auto &emitter = optix_launch_params.emitters[local_hit.emitter_index];
                auto emission = emitter.radiance.Sample(local_hit.geo.texcoord);
                record.radiance += emission;
            }
        }

        ++depth;
        if (depth >= optix_launch_params.config.max_depth)
            break;

        // direct light sampling
        {
            auto &emitter = optix_util::Emitter::SelectOneEmiiter(record.random.Next(), optix_launch_params.emitters);
            auto emitter_local = emitter.SampleDirect(record.random.Next(), record.random.Next());
            float distance = length(emitter_local.position - local_hit.geo.position);
            float3 L = normalize(emitter_local.position - local_hit.geo.position);
            float NoL = dot(local_hit.geo.normal, L);
            float LNoL = dot(emitter_local.normal, -L);

            if (NoL > 0.f && LNoL > 0.f) {
                // shadow ray
                unsigned int occluded = 0u;
                optixTrace(optix_launch_params.handle,
                           local_hit.geo.position, L,
                           0.001f, distance - 0.001f, 0.f,
                           255, OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                           1, 2, 1, occluded);
                if (occluded == 0u) {
                    float light_pdf = distance * distance / (LNoL * emitter.area) * emitter.select_probability;
                    float3 wi = optix_util::ToLocal(L, local_hit.geo.normal);
                    float3 wo = optix_util::ToLocal(-ray_direction, local_hit.geo.normal);
                    auto [f, pdf] = record.hit.mat.Eval(wi, wo, local_hit.geo.texcoord);
                    if (!optix_util::IsZero(f)) {
                        float mis = optix_util::MISWeight(light_pdf, pdf);
                        record.radiance += record.throughput * emitter_local.radiance * f * NoL * mis / light_pdf;
                    }
                }
            }
        }
        // bsdf sampling
        {
            float2 xi = make_float2(record.random.Next(), record.random.Next());
            float3 wo = optix_util::ToLocal(-ray_direction, local_hit.geo.normal);
            auto bsdf_sampled_record = record.hit.mat.Sample(xi, wo, local_hit.geo.texcoord);
            if (optix_util::IsZero(bsdf_sampled_record.f * abs(bsdf_sampled_record.wi.z)) || optix_util::IsZero(bsdf_sampled_record.pdf))
                break;

            record.throughput *= bsdf_sampled_record.f * abs(bsdf_sampled_record.wi.z) / bsdf_sampled_record.pdf;
            float3 sampled_wi = optix_util::ToWorld(bsdf_sampled_record.wi, local_hit.geo.normal);

            double rr = depth > 2 ? 0.95 : 1.0;
            if (record.random.Next() > rr)
                break;
            record.throughput /= rr;
            ray_origin = record.hit.geo.position;
            ray_direction = sampled_wi;

            optixTrace(optix_launch_params.handle,
                       ray_origin, ray_direction,
                       0.001f, 1e16f, 0.f,
                       255, OPTIX_RAY_FLAG_NONE,
                       0, 2, 0,
                       u0, u1);

            local_hit = record.hit;
            float distance = length(ray_origin - local_hit.geo.position);
            if (local_hit.emitter_index >= 0) {
                auto &emitter = optix_launch_params.emitters[local_hit.emitter_index];
                optix_util::Emitter::LocalRecord emitter_local = emitter.GetLocalInfo(local_hit.geo.position);
                float LNoL = dot(emitter_local.normal, -ray_direction);
                if (LNoL > 0.f) {
                    float light_pdf = distance * distance / (LNoL * emitter.area) * emitter.select_probability;
                    float mis = optix_util::MISWeight(bsdf_sampled_record.pdf, light_pdf);
                    if (bsdf_sampled_record.lobe_type & optix_util::EBsdfLobeType::DeltaReflection)
                        mis = 1.f;

                    auto emission = emitter.radiance.Sample(local_hit.geo.texcoord);

                    record.radiance += record.throughput * emission * mis;
                }
            }
        }
    }

    if (optix_launch_params.config.accumulated_flag && optix_launch_params.frame_cnt > 0) {
        const float t = 1.f / (optix_launch_params.frame_cnt + 1.f);
        const float3 pre = make_float3(optix_launch_params.accum_buffer[pixel_index]);
        record.radiance = lerp(pre, record.radiance, t);
    }
    optix_launch_params.accum_buffer[pixel_index] = make_float4(record.radiance, 1.f);

    float3 color = optix_util::ACESToneMapping(record.radiance, 1.f);
    if (optix_launch_params.config.use_tone_mapping)
        color = optix_util::GammaCorrection(color, 2.2f);
    optix_launch_params.frame_buffer[pixel_index] = make_float4(color, 1.f);
}

extern "C" __global__ void __miss__default() {
    auto record = optix_util::GetPRD<PathPayloadRecord>();
    // if (optix_launch_params.env) {
    //     // TODO: environment texture
    //     float2 tex = make_float2(0.f, 0.f);
    //     record->radiance += record->throughput * optix_launch_params.env->Sample(tex);
    // }
    record->done = true;
}
extern "C" __global__ void __miss__shadow() {
    // optixSetPayload_0(0u);
}
extern "C" __global__ void __closesthit__default() {
    const HitGroupData *sbt_data = (HitGroupData *)optixGetSbtDataPointer();
    auto record = optix_util::GetPRD<PathPayloadRecord>();

    const auto ray_dir = optixGetWorldRayDirection();
    const auto ray_o = optixGetWorldRayOrigin();

    record->hit.geo = sbt_data->geo.GetHitLocalGeometry(ray_dir, sbt_data->mat.twosided);
    if (sbt_data->emitter_index_offset >= 0) {
        record->hit.emitter_index = sbt_data->emitter_index_offset + optixGetPrimitiveIndex();
    } else {
        record->hit.emitter_index = -1;
    }

    record->hit.mat = sbt_data->mat;
}
extern "C" __global__ void __closesthit__shadow() {
    optixSetPayload_0(1u);
}